#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>

#include "Animation.h"
#include "GPUFunctions.h"

const uint32_t THREADSPERBLOCK = 1024;
const float forceOfGravity = 500;
const uint32_t AREATOCHECK = 50;
const float maxSpeed = 25;

__global__ void randomizeValues(float * pPosX, float * pPosY, float * pVelX, float * pVelY, uint32_t particleCount, uint32_t width, uint32_t height, uint64_t * rand, uint32_t * color) {
	uint32_t index = blockIdx.x *blockDim.x + threadIdx.x;

	if(index < particleCount) {
		
	}
}

__global__ void nextFrameGPU(float * pPosX, float * pPosY, float * pVelX, float * pVelY, uint32_t particleCount, uint32_t width, uint32_t height, int64_t * occupied, uint32_t * color) {
	uint32_t index = blockIdx.x *blockDim.x + threadIdx.x;

	if(index < particleCount) {
		
	}
}

__global__ void createRender(uint32_t * image, float * pPosX, float * pPosY, uint32_t particleCount, uint32_t width, uint32_t height, uint32_t * color) {
	uint32_t index = blockIdx.x *blockDim.x + threadIdx.x;

	if(index < particleCount) {

	}
}

Animation::Animation(uint32_t widthIn, uint32_t heightIn, uint32_t particlesIn) {
	width = widthIn;
	height = heightIn;
	particleCount = particlesIn;
	// TODO: test if using passed in variables is better than class variables
	particlePositionsX = (float *) gpuMemAlloc(particleCount * sizeof(float));
	particlePositionsY = (float *) gpuMemAlloc(particleCount * sizeof(float));
	particleVelocitiesX = (float *) gpuMemAlloc(particleCount * sizeof(float));
	particleVelocitiesY = (float *) gpuMemAlloc(particleCount * sizeof(float));
	imageSize = width * height * sizeof(uint32_t);
	d_image = (uint32_t *) gpuMemAlloc(imageSize);
	image = (uint32_t *) malloc(imageSize);
	blockCountGPU = (particleCount + THREADSPERBLOCK - 1)/THREADSPERBLOCK;
	init();
}

void Animation::init() {
	randomizeValues <<< blockCountGPU, THREADSPERBLOCK >>> (particlePositionsX, particlePositionsY, particleVelocitiesX,  particleVelocitiesY, particleCount, width, height, d_rand, color);
}

void Animation::nextFrame() {
	hipMemset(d_image, 0x00000000, imageSize);
	nextFrameGPU <<< blockCountGPU, THREADSPERBLOCK >>> (particlePositionsX, particlePositionsY, particleVelocitiesX, particleVelocitiesY, particleCount, width, height, d_occupied, color);
	hipMemset(d_occupied, -1, width * height * sizeof(int64_t));
	createRender <<< blockCountGPU, THREADSPERBLOCK >>> (d_image, particlePositionsX, particlePositionsY, particleCount, width, height, color);
}

void Animation::exit() {
	hipFree(d_image);
	hipFree(particlePositionsX);
	hipFree(particlePositionsY);
	hipFree(particleVelocitiesX);
	hipFree(particleVelocitiesY);
}

void * Animation::getImage() {
	hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost);
	return image;
}